// Copyright 2016 Caoyang Jiang


#include <hip/hip_runtime.h>
#include <cstdint>
#include <iostream>
#include <vector>

#define NNEIGHBOR 10

__device__ static bool CUDAFuncIsValInArray(int* array, int len, int val)
{
  for (int i = 0; i < len; i++)
  {
    if (array[i] == val) return true;
  }

  return false;
}
__global__ static void CUDAKernelFindNeighbor(size_t pointcnt,
                                              size_t polygoncnt,
                                              uint32_t* v1,
                                              uint32_t* v2,
                                              uint32_t* v3,
                                              int (*connectivity)[NNEIGHBOR])
{
  int neighbor[NNEIGHBOR];
  int neighborcnt = 0;
  int it          = 0;
  const int x     = blockIdx.x * blockDim.x + threadIdx.x;
  const int mx    = gridDim.x * blockDim.x;

  for (int in = 0; in < NNEIGHBOR; in++)
  {
    neighbor[in] = -1;
  }

  while ((x + it * mx) < pointcnt)
  {
    uint32_t idx = x + it * mx;

    for (size_t polysz = 0; polysz < polygoncnt; polysz++)
    {
      uint32_t v[3];

      v[0] = v1[polysz];
      v[1] = v2[polysz];
      v[2] = v3[polysz];

      if (idx == v[0])
      {
        if (!CUDAFuncIsValInArray(neighbor, NNEIGHBOR, v[1]))
        {
          neighbor[neighborcnt] = v[1];
          neighborcnt++;
        }

        if (!CUDAFuncIsValInArray(neighbor, NNEIGHBOR, v[2]))
        {
          neighbor[neighborcnt] = v[2];
          neighborcnt++;
        }
      }

      if (idx == v[1])
      {
        if (!CUDAFuncIsValInArray(neighbor, NNEIGHBOR, v[0]))
        {
          neighbor[neighborcnt] = v[0];
          neighborcnt++;
        }

        if (!CUDAFuncIsValInArray(neighbor, NNEIGHBOR, v[2]))
        {
          neighbor[neighborcnt] = v[2];
          neighborcnt++;
        }
      }

      if (idx == v[2])
      {
        if (!CUDAFuncIsValInArray(neighbor, NNEIGHBOR, v[0]))
        {
          neighbor[neighborcnt] = v[0];
          neighborcnt++;
        }

        if (!CUDAFuncIsValInArray(neighbor, NNEIGHBOR, v[1]))
        {
          neighbor[neighborcnt] = v[1];
          neighborcnt++;
        }
      }
    }

    for (int in = 0; in < NNEIGHBOR; in++)
    {
      connectivity[idx][in] = neighbor[in];
      neighbor[in]          = -1;
    }
    // std::sort(connectivity[sz].begin(), connectivity[sz].end(),
    // CompS2LU32);
    it++;
    neighborcnt = 0;
  }
}

bool FindNeighbor(int pointcnt,
                  const std::vector<std::vector<uint32_t>>& polygons,
                  std::vector<std::vector<uint32_t>>& connectivity)
{
  uint32_t* data[3];

  int(*cudaconnectivity)[NNEIGHBOR];

  hipMallocManaged(reinterpret_cast<void**>(&data[0]),
                    polygons.size() * sizeof(uint32_t),
                    hipMemAttachGlobal);
  hipMallocManaged(reinterpret_cast<void**>(&data[1]),
                    polygons.size() * sizeof(uint32_t),
                    hipMemAttachGlobal);
  hipMallocManaged(reinterpret_cast<void**>(&data[2]),
                    polygons.size() * sizeof(uint32_t),
                    hipMemAttachGlobal);
  hipMallocManaged(reinterpret_cast<void**>(&cudaconnectivity),
                    pointcnt * sizeof(int[NNEIGHBOR]),
                    hipMemAttachGlobal);

  std::cout << polygons.size() << std::endl;
  for (size_t sz = 0; sz < polygons.size(); sz++)
  {
    data[0][sz] = polygons[sz][0];
    data[1][sz] = polygons[sz][1];
    data[2][sz] = polygons[sz][2];
  }

  dim3 blks(64, 1);
  dim3 threads(256, 1);

  CUDAKernelFindNeighbor<<<blks, threads>>>(
      pointcnt, polygons.size(), data[0], data[1], data[2], cudaconnectivity);

  hipDeviceSynchronize();

  for (int ip = 0; ip < pointcnt; ip++)
  {
    std::vector<uint32_t> tmp;
    for (int in = 0; in < NNEIGHBOR; in++)
    {
      if (cudaconnectivity[ip][in] != -1)
      {
        tmp.push_back(cudaconnectivity[ip][in]);
      }
    }
    connectivity.push_back(tmp);
  }

  hipFree(data[0]);
  hipFree(data[1]);
  hipFree(data[2]);
  hipFree(cudaconnectivity);

  return true;
}
