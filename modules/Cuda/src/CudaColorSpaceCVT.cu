#include "hip/hip_runtime.h"
// Copyright 2016 Caoyang Jiang

#include <hip/hip_runtime.h>
#include "Jcy/Cuda/CudaColorSpaceCVT.h"

__global__ static void CudaKernelYuv420ToRgb(int* data)
{
  const int x  = blockIdx.x * blockDim.x + threadIdx.x;
  const int y  = blockIdx.y * blockDim.y + threadIdx.y;
  const int mx = gridDim.x * blockDim.x;

  data[y * mx + x] = data[y * mx + x] + 1.0f;
}

namespace jcy
{
bool CudaColorSpaceCVT::CpuConvert(const uint8_t* in)
{
  return true;
}
bool CudaColorSpaceCVT::GpuConvert(const uint8_t* in)
{
  hipError_t error;
  if (incs_ == ColorSpace::YUV420)
  {
    // Copy first component
    if ((error = hipMemcpy2D(reinterpret_cast<void*>(incompptr_[0]),
                              static_cast<size_t>(kMAXSIZE),
                              reinterpret_cast<const void*>(in),
                              static_cast<size_t>(width_),
                              static_cast<size_t>(width_),
                              static_cast<size_t>(height_),
                              hipMemcpyHostToDevice)) != hipSuccess)
    {
      std::cout << "[ERROR]: Copy input first component onto GPU failed, "
                << hipGetErrorString(error) << std::endl;
      return false;
    }

    // Copy Second component
    if ((error =
             hipMemcpy2D(reinterpret_cast<void*>(incompptr_[1]),
                          static_cast<size_t>(kMAXSIZE),
                          reinterpret_cast<const void*>(in + width_ * height_),
                          static_cast<size_t>(width_ / 2),
                          static_cast<size_t>(width_ / 2),
                          static_cast<size_t>(height_ / 2),
                          hipMemcpyHostToDevice)) != hipSuccess)
    {
      std::cout << "[ERROR]: Copy input second component onto GPU failed, "
                << hipGetErrorString(error) << std::endl;
      return false;
    }

    // Copy Third component
    if ((error = hipMemcpy2D(
             reinterpret_cast<void*>(incompptr_[2]),
             static_cast<size_t>(kMAXSIZE),
             reinterpret_cast<const void*>(in + width_ * height_ * 5 / 4),
             static_cast<size_t>(width_ / 2),
             static_cast<size_t>(width_ / 2),
             static_cast<size_t>(height_ / 2),
             hipMemcpyHostToDevice)) != hipSuccess)
    {
      std::cout << "[ERROR]: Copy input third component onto GPU failed, "
                << hipGetErrorString(error) << std::endl;
      return false;
    }
  }
  else if (incs_ == ColorSpace::RGB444)
  {
    for (int icomp = 0; icomp < 3; icomp++)
    {
      // Copy first component
      if ((error = hipMemcpy2D(
               reinterpret_cast<void*>(incompptr_[icomp]),
               static_cast<size_t>(kMAXSIZE),
               reinterpret_cast<const void*>(in + icomp * width_ * height_),
               static_cast<size_t>(width_),
               static_cast<size_t>(width_),
               static_cast<size_t>(height_),
               hipMemcpyHostToDevice)) != hipSuccess)
      {
        std::cout << "[ERROR]: Copy input " << icomp
                  << " component onto GPU failed, " << hipGetErrorString(error)
                  << std::endl;
        return false;
      }
    }
  }
  else
  {
    std::cout << "[ERROR]: Unsupported input color space." << std::endl;
    return false;
  }

  //
  return true;
}

CudaColorSpaceCVT::CudaColorSpaceCVT()
{
  hipError_t error;

  if ((error = hipMallocManaged(reinterpret_cast<void**>(&inbuffer),
                                 kMAXSIZE * kMAXSIZE * kMAXCOMP)) !=
      hipSuccess)
  {
    std::cout << "[ERROR]: Malloc buffer failed, " << hipGetErrorString(error)
              << std::endl;
    exit(1);
  }

  if ((error = hipMallocManaged(reinterpret_cast<void**>(&cvtbuffer),
                                 kMAXSIZE * kMAXSIZE * kMAXCOMP)) !=
      hipSuccess)
  {
    std::cout << "[ERROR]: Malloc buffer failed, " << hipGetErrorString(error)
              << std::endl;
    exit(1);
  }

  // For conenience purpose.
  incompptr_[0] = inbuffer;
  incompptr_[1] = inbuffer + kMAXSIZE;
  incompptr_[2] = inbuffer + 2 * kMAXSIZE;

  cvtcompptr_[0] = cvtbuffer;
  cvtcompptr_[1] = cvtbuffer + kMAXSIZE;
  cvtcompptr_[2] = cvtbuffer + 2 * kMAXSIZE;
}

CudaColorSpaceCVT::~CudaColorSpaceCVT()
{
}

void CudaColorSpaceCVT::SetWidth(int width)
{
  width_ = width;
}

void CudaColorSpaceCVT::SetHeight(int height)
{
  height_ = height;
}

void CudaColorSpaceCVT::SetHardware(int hardware)
{
  hardware_ = hardware;
}

void CudaColorSpaceCVT::SetInOutCS(enum ColorSpace incs, enum ColorSpace outcs)
{
  incs_  = incs;
  outcs_ = outcs;
}

bool CudaColorSpaceCVT::Convert(const uint8_t* fb)
{
  if (hardware_ == 0)
  {
    return CpuConvert(fb);
  }
  else if (hardware_ == 1)
  {
    return GpuConvert(fb);
  }
  else
  {
    std::cout << "[ERROR]: Unsupported hardware type " << hardware_
              << std::endl;
    return false;
  }
}

const uint8_t* CudaColorSpaceCVT::GetConvertedFrm() const
{
  return cvtbuffer;
}

}  // namespace jcy
