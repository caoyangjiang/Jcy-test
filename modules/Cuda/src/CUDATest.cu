#include "hip/hip_runtime.h"
// Copyright 2015 Jason Juang

#include "Jcy/Cuda/CUDATest.cuh"

__global__ static void CUDAKernelAddOneToArray(int *data)
{
  const int x  = blockIdx.x * blockDim.x + threadIdx.x;
  const int y  = blockIdx.y * blockDim.y + threadIdx.y;
  const int mx = gridDim.x * blockDim.x;

  data[y * mx + x] = data[y * mx + x] + 1.0f;
}

namespace jcy
{
CUDATest::CUDATest()
{
}

CUDATest::~CUDATest()
{
}

void CUDATest::AddOneToArray(std::vector<int> &in)
{
  if (in.empty()) return;

  cudaconfig_.w_a = static_cast<int>(in.size()) / cudaconfig_.blk_h;
  cudaconfig_.h_a = cudaconfig_.blk_h;

  hipMallocManaged(reinterpret_cast<void **>(&data_),
                    in.size() * sizeof(int),
                    hipMemAttachGlobal);

  for (int i = 0; i < in.size(); i++)
  {
    data_[i] = in.at(i);
  }

  dim3 blks((cudaconfig_.w_a >> cudaconfig_.shift_w),
            (cudaconfig_.h_a >> cudaconfig_.shift_h));
  dim3 threads(cudaconfig_.blk_w, cudaconfig_.blk_h);

  CUDAKernelAddOneToArray<<<blks, threads>>>(data_);

  hipDeviceSynchronize();

  for (int i = 0; i < in.size(); i++)
  {
    in.at(i) = data_[i];
  }

  hipFree(data_);
}

}  // namespace jcy
